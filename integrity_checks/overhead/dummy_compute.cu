#define GNU_SOURCE

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define CUDA_CALL(x) { checkCuda((x), __FILE__, __LINE__); }

inline void checkCuda(hipError_t code, const char * file, int line){
	if (code != hipSuccess){
		fprintf(stderr, "CUDA Error in File %s, Line %d: %s\n", file, line, hipGetErrorString(code));
		exit(1);
	}
}

__global__ void dummyComputeKernel(size_t loop_bound){
	int ind = blockDim.x * blockIdx.x + threadIdx.x;
	// USE "VOLATILE" so compiler does not optimize away (still doesn't work!)
	// using -g -G flags in Makefile to ensure loop is executed...
	for (volatile int i = 0; i < loop_bound; i++) {
		// dummy loop
	}
}

int main(int argc, char *argv[]){
	
	// get device info so we can know how many SMs	
	int device;
	CUDA_CALL(hipGetDevice(&device));

	struct hipDeviceProp_t devProp;
	CUDA_CALL(hipGetDeviceProperties(&devProp, device));

	int sm_count = devProp.multiProcessorCount;
	int max_thread_per_block = devProp.maxThreadsPerBlock;
	
	char * env_with_monitor = getenv("WITH_MONITOR");
	char * env_num_cpus = getenv("NUM_CPUS");

	if ((env_with_monitor == NULL) || (env_num_cpus == NULL)) {
		fprintf(stderr, "ERROR: Usage. Must declare 'WITH_MONITOR' & 'NUM_CPUS' env. variables\n");
		exit(1);
	}

	int is_with_monitoring = atoi(env_with_monitor);
	int num_cpus = atoi(env_num_cpus);

	char * hostbuffer = (char *) malloc(256 * sizeof(char));
	int hostname_ret = gethostname(hostbuffer, 256);
	if (hostname_ret == -1){
		fprintf(stderr, "Could not get hostname, exiting...\n");
		exit(1);
	}
	
	FILE * complete_timing_out_file;
	if (is_with_monitoring == 1){
		complete_timing_out_file = fopen("with_monitoring_dummy_compute.csv", "a");
	}
	else {
		complete_timing_out_file = fopen("raw_dummy_compute.csv", "a");
	}
        
	
	// set the device to have 70GB to identify which device on node we are working on
	// + don't get docked for Slurm priority for under-use
        size_t gb = 1024 * 1024 * 1024;
        float * d_p;
        CUDA_CALL(hipMalloc(&d_p, 70 * gb));

	
	int blocks = sm_count;
	int threads = max_thread_per_block;

	dim3 gridDimDummy(blocks);
	dim3 blockDimDummy(threads);
	
	size_t dummy_loop_bound = 1e8;
	
	struct timespec start, stop;
	uint64_t timestamp_start, timestamp_stop, elapsed;
	
	printf("Launching kernel...\n");

	clock_gettime(CLOCK_REALTIME, &start);
	timestamp_start = start.tv_sec * 1e9 + start.tv_nsec;

	// ACTUALLY LAUNCH KERNEL
	dummyComputeKernel <<< gridDimDummy, blockDimDummy >>> (dummy_loop_bound);
	
	// non-blocking cpu-side so must wait to kernel to finish before recording timing
	hipDeviceSynchronize();

	clock_gettime(CLOCK_REALTIME, &stop);
	timestamp_stop = stop.tv_sec * 1e9 + stop.tv_nsec;
	elapsed = timestamp_stop - timestamp_start;

	fprintf(complete_timing_out_file, "%ld,%ld,%ld,%s,%d,%ld\n", elapsed, timestamp_start, timestamp_stop, hostbuffer, num_cpus, dummy_loop_bound);

	CUDA_CALL(hipFree(d_p));	
}
