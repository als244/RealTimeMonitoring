#define GNU_SOURCE

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define CUDA_CALL(x) { checkCuda((x), __FILE__, __LINE__); }

inline void checkCuda(hipError_t code, const char * file, int line){
	if (code != hipSuccess){
		fprintf(stderr, "CUDA Error in File %s, Line %d: %s\n", file, line, hipGetErrorString(code));
		exit(1);
	}
}

__global__ void dummyComputeKernel(size_t loop_bound){
	int ind = blockDim.x * blockIdx.x + threadIdx.x;
	// USE "VOLATILE" so compiler does not optimize away (confirmed with compiler explorer tool)
	for (volatile int i = 0; i < loop_bound; i++) {
		// dummy loop
	}
}

int main(int argc, char *argv[]){
	
	// get device info so we can know how many SMs	
	int device;
	CUDA_CALL(hipGetDevice(&device));

	struct hipDeviceProp_t devProp;
	CUDA_CALL(hipGetDeviceProperties(&devProp, device));

	int sm_count = devProp.multiProcessorCount;
	int max_thread_per_block = devProp.maxThreadsPerBlock;
	int max_thread_per_sm = devProp.maxThreadsPerMultiProcessor;
	

	// info for dumping and matching with monitoring data
	struct timespec time;

	int is_with_monitoring = atoi(getenv("WITH_MONITORING"));
	int num_cpus = atoi(getenv("NUM_CPUS"));

	char * hostbuffer = (char *) malloc(256 * sizeof(char));
	int hostname_ret = gethostname(hostbuffer, 256);
	if (hostname_ret == -1){
		fprintf(stderr, "Could not get hostname, exiting...\n");
		exit(1);
	}
	
	FILE * compute_test_out_file;
	FILE * complete_timing_out_file;
	if (is_with_monitoring == 1){
		compute_test_out_file = fopen("with_monitoring_computeTest.csv", "a");
		complete_timing_out_file = fopen("with_monitoring_simple_compute.csv", "a");
	}
	else {
		compute_test_out_file = fopen("raw_computeTest.csv", "a");
		complete_timing_out_file = fopen("raw_simple_compute.csv", "a");
	}
        
	
	// sleep for 5 seconds between kernel launches
	size_t micros_sleep = 5 * 1e6;


	// set the device to have 70GB to identify which device on node we are working on
        size_t gb = 1024 * 1024 * 1024;
        float * d_p;
        CUDA_CALL(hipMalloc(&d_p, 70 * gb));

	
	int n_block_sizes = 27;
	int blockSizes[] = {1, 2, 4, 8, 16, 32, 64, 70, 75, 80, 81, 82, 83, 84, 85, 86, 100, 116, 128, 168, 252, 256, 336, 512, 672, 840, 1024};

	int n_thread_sizes = 11;
	int threadSizes[] = {1, 2, 4, 8, 16, 32, 64, 128, 256, 512, 1024};
	
	int blocks, threads;
	
	// every thread iterating to 1 billion 
	// (arbritrary number, need large enough to see measurements from kernel in monitoring, but not too large that it is wasting time...)
	// Will look at FieldIds 203 (normal GPU utilization), 1002 (SM active = ratio of cycles 1 warp scheduled), 1003 (SM occupancy = ratio of warps resident on SM to theoretical maximum)
	// hoping to see that with less blocks launched that 203 & 1002 show high numbmers and 1003 is low. As GPU gets saturated expect 1003 to increase...
	size_t dummy_loop_bound = 1e8;
	
	struct timespec start, stop;
	uint64_t timestamp_start, timestamp_stop, elapsed;

	clock_gettime(CLOCK_REALTIME, &start);
	timestamp_start = start.tv_sec * 1e9 + start.tv_nsec;

	for (int blockInd = 0; blockInd < n_block_sizes; blockInd++){
		for (int threadInd = 0; threadInd < n_thread_sizes; threadInd++){
			blocks = blockSizes[blockInd];
			threads = threadSizes[threadInd];
			dim3 gridDimDummy(blocks);
			dim3 blockDimDummy(threads);
			clock_gettime(CLOCK_REALTIME, &time);
			// Current time (ns), about to launch #blocks, about to launch #threads
			fprintf(compute_test_out_file, "%ld,%d,%d\n", time.tv_sec * 1e9 + time.tv_nsec, blocks, threads);
			fflush(compute_test_out_file);
			dummyComputeKernel <<< gridDimDummy, blockDimDummy >>> (dummy_loop_bound);
			hipDeviceSynchronize();
			usleep(micros_sleep);
		}
	}

	clock_gettime(CLOCK_REALTIME, &stop);
	timestamp_stop = stop.tv_sec * 1e9 + stop.tv_nsec;
	elapsed = timestamp_stop - timestamp_start;
	
	fprintf(complete_timing_out_file, "%ld,%ld,%ld,%s,%d\n", elapsed, timestamp_start, timestamp_stop, hostbuffer, num_cpus);

	CUDA_CALL(hipFree(d_p));	
}
